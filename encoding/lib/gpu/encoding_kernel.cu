#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/serialize/tensor.h>

#include "common.h"
#include "device_tensor.h"

namespace {

template<typename DType, typename Acctype>
struct AggOp {
  __device__ AggOp(DeviceTensor<DType, 3> a,
                   DeviceTensor<DType, 3> x,
                   DeviceTensor<DType, 2> c) : A(a), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(A[b][i][k] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> A;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
struct AggBackOp {
  __device__ AggBackOp(DeviceTensor<DType, 3> g,
                       DeviceTensor<DType, 3> x,
                       DeviceTensor<DType, 2> c) : G(g), X(x), C(c) {}
  __device__ __forceinline__ Acctype operator()(int b, int i, int k, int d) {
    return ScalarConvert<DType, Acctype>::to(G[b][k][d] * (X[b][i][d] - C[k][d]));
  }
  DeviceTensor<DType, 3> G;
  DeviceTensor<DType, 3> X;
  DeviceTensor<DType, 2> C;
};

template<typename DType, typename Acctype>
__global__ void Aggregate_Forward_kernel (
    DeviceTensor<DType, 3> E,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, d, N;
  /* Get the index and channels */ 
  b = blockIdx.z;
  d = blockIdx.x;
  k = blockIdx.y;
  N = X.getSize(1);
  /* main operation */
  AggOp<DType, Acctype> g(A, X, C);
  E[b][k][d] = reduceN<Acctype>(g, b, k, d, N);
}

template<typename DType, typename Acctype>
__global__ void Aggregate_Backward_kernel (
    DeviceTensor<DType, 3> GA,
    DeviceTensor<DType, 3> GE,
    DeviceTensor<DType, 3> A,
    DeviceTensor<DType, 3> X,
    DeviceTensor<DType, 2> C) {
  /* declarations of the variables */
  int b, k, i, D;
  /* Get the index and channels */ 
  b = blockIdx.z;
  i = blockIdx.y;
  k = blockIdx.x;
  D = GE.getSize(2);
  /* main operation */
  AggBackOp<DType, Acctype> g(GE, X, C);
  GA[b][i][k] = reduceD<Acctype>(g, b, i, k, D);
}

}// namespace

at::Tensor Aggregate_Forward_CUDA(
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  /* Device tensors */
  auto E_ = torch::zeros({A_.size(0), C_.size(0), C_.size(1)}, A_.options());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(1), C_.size(0), X_.size(0));
  dim3 threads(getNumThreads(X_.size(1)));

  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Forward_CUDA", ([&] {
    DeviceTensor<scalar_t, 3> E = devicetensor<scalar_t, 3>(E_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    /* kernel function */
    Aggregate_Forward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>>(E, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return E_;
}

std::vector<at::Tensor> Aggregate_Backward_CUDA(
    const at::Tensor GE_,
    const at::Tensor A_,
    const at::Tensor X_,
    const at::Tensor C_) {
  auto gradA_ = at::zeros_like(A_);
  auto gradX_ = at::bmm(A_, GE_);
  auto gradC_ = (-GE_ * A_.sum(1).unsqueeze(2)).sum(0);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // B, K, D
  dim3 blocks(C_.size(0), X_.size(1), X_.size(0));
  dim3 threads(getNumThreads(C_.size(1)));
  AT_DISPATCH_FLOATING_TYPES(A_.type(), "Aggregate_Backward_CUDA", ([&] {
    /* Device tensors */
    DeviceTensor<scalar_t, 3> GA = devicetensor<scalar_t, 3>(gradA_);
    DeviceTensor<scalar_t, 3> GE = devicetensor<scalar_t, 3>(GE_);
    DeviceTensor<scalar_t, 3> A = devicetensor<scalar_t, 3>(A_);
    DeviceTensor<scalar_t, 3> X = devicetensor<scalar_t, 3>(X_);
    DeviceTensor<scalar_t, 2> C = devicetensor<scalar_t, 2>(C_);
    Aggregate_Backward_kernel<scalar_t, scalar_t>
      <<<blocks, threads, 0, stream>>> (GA, GE, A, X, C);
  }));
  AT_ASSERT(hipGetLastError() == hipSuccess);
  return {gradA_, gradX_, gradC_};
}
